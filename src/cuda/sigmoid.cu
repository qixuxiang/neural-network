#include "hip/hip_runtime.h"
#include <sigmoid.cuh>

struct sigmoid_functor {
  __host__ __device__ float operator()(const float &x) const {
    return 1 / (1 + expf(-x));
  }
};

struct sigmoid_d_functor {
  __host__ __device__ float operator()(const float &x) const {
    float s = 1 / (1 + expf(-x));
    return s * (1 - s);
  }
};

void operator_sigmoid(const Storage *input1, Storage *output) {
  thrust::transform(input1->get_data().begin(), input1->get_data().end(),
                    output->get_data().begin(), sigmoid_functor());
}

// Y = sigmoid(X)
// dL/dX = sigmoid'(X) element_mul dL/dY
void operator_d_sigmoid(
    const Storage *outputs_grad, const Storage *input1, Storage *inputs_grad,
    std::unordered_map<std::string, std::unique_ptr<Storage>> &temp) {
  INIT_TEMP(temp, "d_sigmoid", input1->get_shape());
  thrust::transform(input1->get_data().begin(), input1->get_data().end(),
                    temp["d_sigmoid"]->get_data().begin(), sigmoid_d_functor());

  operator_mul(temp["d_sigmoid"].get(), outputs_grad, inputs_grad);
}

void Sigmoid::forward() {
  const Storage *input = this->pre->get_output();

  INIT_STORAGE(this->output, input->get_shape());
  operator_sigmoid(input, this->output.get());
}

void Sigmoid::backward() {
  const Storage *input = this->pre->get_output();
  const Storage *output_grad = this->next->get_grad();

  INIT_STORAGE(this->grad, input->get_shape());
  operator_d_sigmoid(output_grad, input, this->grad.get(), this->temp);
}
